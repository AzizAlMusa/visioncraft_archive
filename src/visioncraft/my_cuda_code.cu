#include <hip/hip_runtime.h>
#include <iostream>

// CUDA kernel definition
__global__ void advancedCudaKernel(float* data, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        data[idx] = data[idx] * data[idx] + 2.0f; // More complex operation
    }
}

// CPU computation function
void cpuComputation(float* data, int size) {
    for (int i = 0; i < size; ++i) {
        data[i] = data[i] * data[i] + 2.0f; // Same operation as GPU for comparison
    }
}

// GPU computation function with timing
void run_advanced_cuda_example(int size) {
    float* d_data;
    float* h_data = new float[size];

    // Allocate memory on the GPU
    hipMalloc(&d_data, size * sizeof(float));
    hipMemset(d_data, 1, size * sizeof(float)); // Initialize to 1

    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record start time
    hipEventRecord(start);

    // Define block and grid sizes
    int blockSize = 256;
    int numBlocks = (size + blockSize - 1) / blockSize;

    // Launch the CUDA kernel
    advancedCudaKernel<<<numBlocks, blockSize>>>(d_data, size);

    // Record stop time
    hipEventRecord(stop);

    // Wait for the event to complete
    hipEventSynchronize(stop);

    // Calculate the elapsed time
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    // Free the GPU memory
    hipFree(d_data);

    // Clean up events
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    delete[] h_data;

    std::cout << "CUDA kernel execution time: " << elapsedTime / 1000.0 << " seconds" << std::endl; // Convert to seconds
}
